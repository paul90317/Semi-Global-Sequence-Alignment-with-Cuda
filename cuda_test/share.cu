#include "hip/hip_runtime.h"

#include <iostream>
#include "test_time.h"
#include "utils.cuh"

#define THREADS 64
#define BLOCKS 2048
#define TIME 300000
#define COUNT (THREADS*BLOCKS)

__global__ void sadder(int* a,int *b,int *c){
    int gid=TID;
    int tid=threadIdx.x;
    __shared__ static int sb[THREAD_SIZE];
    __shared__ static int sc[THREAD_SIZE];
    sb[tid]=b[gid];
    sc[tid]=c[gid];
    a[gid]=sb[tid]*sc[tid];
}
__global__ void adder(int* a,int *b,int *c){
    int tid=TID;
    a[tid]=b[tid]*c[tid]+c[tid]*(b[tid]+1);
}
__global__ void loop_adder(int* a,int *b,int *c){
    int tid=TID;
    for(int i=0;i<TIME;i++){
        a[tid]+=b[tid]+c[tid];
    }
}

int main(){
    int *a,*b,*c;
    hipMalloc(&a,sizeof(int)*COUNT);
    hipMalloc(&b,sizeof(int)*COUNT);
    hipMalloc(&c,sizeof(int)*COUNT);
    time_start();
    for(int i=0;i<TIME;i++){
        sadder _kernel(BLOCKS,THREADS)(a,b,c);
    }
    //loop_adder _kernel(BLOCKS,THREADS)(a,b,c);
    time_end();
}