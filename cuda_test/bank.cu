#include "hip/hip_runtime.h"

#include <iostream>
#include "test_time.h"
#include "utils.cuh"

#define THREADS 64
#define BLOCKS 2048
#define TIME 300000
#define COUNT (THREADS*BLOCKS)

__global__ void adder1(){//bank conflict
    int gid=TID;
    int tid=threadIdx.x;
    __shared__ static int m[THREAD_SIZE][4];
    m[tid][0]=m[tid][1]+m[tid][2]+m[tid][3];
}
__global__ void adder2(){//no bank conflict, but not continue
    int gid=TID;
    int tid=threadIdx.x;
    __shared__ static int m[4][THREAD_SIZE];
    m[0][tid]=m[1][tid]+m[2][tid]+m[3][tid];
}
__global__ void adder3(){//no bank conflict
    int gid=TID;
    int tid=threadIdx.x;
    __shared__ static int m[THREAD_SIZE][5];
    m[tid][0]=m[tid][1]+m[tid][2]+m[tid][3];
}
int main(){
    time_start();
    for(int i=0;i<TIME;i++){
        adder3 _kernel(BLOCKS,THREADS)();
    }
    time_end();
}