#include "hip/hip_runtime.h"

#include <iostream>
#include "test_time.h"
#include "utils.cuh"

#define COUNT 1000000
#define TIME 100000
//#define dimcf(i,j) ((i)+(j)*COUNT)
#define dimcf(i,j) (3*(i)+(j))
__global__ void _cal(int* M,int count){
    int tid=TID;
    if(tid>=count)return;
    M[dimcf(tid,0)]=M[dimcf(tid,1)]+M[dimcf(tid,2)];
}
void cal(int* M,int count){
    int nb,nt;
    thread_assign(count,&nb,&nt);
    _cal _kernel(nb,nt)(M,count);
}

int main(){
    int *M;
    hipMalloc(&M,sizeof(int)*COUNT*3);
    time_start();
    for(int i=0;i<TIME;i++){
        cal(M,COUNT);
    }
    time_end();
}