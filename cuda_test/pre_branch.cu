#include "hip/hip_runtime.h"

#include <iostream>
#include "test_time.h"
#include "utils.cuh"

#define COUNT 1000000
#define TIME 100000
#define PRE_BRANCH (COUNT/3)
//這是為了要確定如果先 return 就不需要進 warp
__global__ void _move(int* a,int *b,int count){
    int tid=TID;
    if(tid>=PRE_BRANCH)return;
    a[tid]=b[tid];
}
void move(int* a,int* b,int count){
    int nb,nt;
    thread_assign(count,&nb,&nt);
    _move _kernel(nb,nt)(a,b,count);
}

int main(){
    int *a,*b;
    hipMalloc(&a,sizeof(int)*COUNT);
    hipMalloc(&b,sizeof(int)*COUNT);
    time_start();
    for(int i=0;i<TIME;i++){
        move(a,b,COUNT);
    }
    time_end();
}