

#include <hip/hip_runtime.h>
#include <cstdio>

#define NUM_BLOCKS 64
#define NUM_THREADS 256
__global__ void f()
{
    int tid=gridDim.x;
    printf("%d ",tid);
}
int main(){
    int gpu_n;
    hipGetDeviceCount(&gpu_n);
    printf("CUDA-capable device count: %i\n", gpu_n);
    f<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 *NUM_THREADS>>>();
}