#include "hip/hip_runtime.h"

#include <iostream>
#include "test_time.h"
#include "utils.cuh"

#define THREADS 64
#define BLOCKS 2048
#define TIME 300000
#define COUNT (THREADS*BLOCKS)
__global__ void adder(int* a,int *b,int *c){
    int tid=TID;
    a[tid]+=b[tid]+c[tid];
}

__global__ void loop_adder(int* a,int *b,int *c){
    int tid=TID;
    for(int i=0;i<TIME;i++){
        a[tid]+=b[tid]+c[tid];
    }
}

int main(){
    int *a,*b,*c;
    hipMalloc(&a,sizeof(int)*COUNT);
    hipMalloc(&b,sizeof(int)*COUNT);
    hipMalloc(&c,sizeof(int)*COUNT);
    time_start();
    for(int i=0;i<TIME;i++){
        adder _kernel(BLOCKS,THREADS)(a,b,c);
    }
    //loop_adder _kernel(BLOCKS,THREADS)(a,b,c);
    time_end();
}