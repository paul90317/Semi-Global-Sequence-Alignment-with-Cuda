#ifndef UTILS
#define UTILS

template<typename T>
inline void gmalloc(T** arr,int sz){
    hipMalloc(arr, (sz)*sizeof(T));
}

template<typename T>
inline void gmemcpy_h2d(T* dst, T* src, int sz){
    hipMemcpy(dst, src, sz*sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
inline void gmemcpy_d2d(T* dst, T* src, int sz){
    hipMemcpy(dst, src, sz*sizeof(T), hipMemcpyDeviceToDevice);
}

template<typename T>
inline void gmemcpy_d2h(T* dst, T* src, int sz){
    hipMemcpy(dst, src, sz*sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
inline void gmemset(T* arr, char val, int sz){
    hipMemset(arr, val, sz*sizeof(T));
}

template<typename T>
inline void gassign(T* p,T val){
    gmemcpy_h2d(p,&val,sizeof(T));
}
#endif