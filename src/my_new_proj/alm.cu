#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>
#include "afg_controller.cu"

#define THREAD_SIZE 1024

using namespace std;

__global__ void f(afg_unit** g,int sz){
    afg_unit* tmp=*g;
    printf("bbb\n");
    for(int i=0;i<sz;i++){
        printf("%d %d %d\n",tmp[i].m.score,tmp[i].x.score,tmp[i].y.score);
    }
}
__global__ void g(){
    printf("ccc\n");
    /*for(int i=0;i<sz;i++){
        printf("%d %d %d\n",tmp[i].m.score,tmp[i].x.score,tmp[i].y.score);
    }*/
}
int main(int argc,char** argv){
    
    char*x="123";
    char*y="12345";
    afg_controller ac=afg_controller(x,y,3,5,0);
    printf("aaa\n");
    f<<<1,1>>>(ac.GM,(ac.xsize+2)*3);
    g<<<1,1>>>();
    
}

