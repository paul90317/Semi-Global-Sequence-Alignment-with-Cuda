#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>

#include "utils.cuh"
#include "afg_unit.cuh"
#include "config.cuh"
#include "layout.cuh"
#define Y_NOT_END(idy,xsize,ysize) (((idy)-(xsize)-1)<=(ysize))

__global__ void calculate_yfreeStart(afg_unit* M,int* x,int* y,int index_y,res_unit* gbest,int xsize,int ysize) {
    int tid=TID;
    int xid=tid+1;
    int yid=index_y-xid-1;
    if(xid<1||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(xid==1){
        M[dimcf(0,xid)].x.ystart=yid+1;
        M[dimcf(0,xid)].m.ystart=yid;
    }
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END||Y_FREE_END&&X_FREE_END){
        M[dimcf(0,xid)].result().xend=xid;
        M[dimcf(0,xid)].result().yend=yid;
        *gbest=max2(*gbest,M[dimcf(0,xid)].result());
    }
}
__global__ void calculate_xyfreeStart(afg_unit* M,int* x,int* y,int index_y,res_unit* gbest,int xsize,int ysize) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    res_unit zero(0,xid+1,xid,yid+1,yid,false);
    M[dimcf(0,xid)].x=max2(M[dimcf(1,xid-1)].to_x(),zero);
    M[dimcf(0,xid)].y=max2(M[dimcf(1,xid)].to_y(),zero);
    M[dimcf(0,xid)].m=max2(M[dimcf(2,xid-1)].to_m(x[xid],y[yid]),zero);
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END||Y_FREE_END&&X_FREE_END){
        M[dimcf(0,xid)].result().xend=xid;
        M[dimcf(0,xid)].result().yend=yid;
        *gbest=max2(*gbest,M[dimcf(0,xid)].result());
    }
}
__global__ void calculate_fixedStart(afg_unit* M,int* x,int* y,int index_y,res_unit* gbest,int xsize,int ysize) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END||Y_FREE_END&&X_FREE_END){
        M[dimcf(0,xid)].result().xend=xid;
        M[dimcf(0,xid)].result().yend=yid;
        *gbest=max2(*gbest,M[dimcf(0,xid)].result());
    }
}
__global__ void calculate_xfreeStart(afg_unit* M,int* x,int* y,int index_y,res_unit* gbest,int xsize,int ysize) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(yid==0){
        M[dimcf(0,xid)].m=0;
    }
    if(yid==1){
        M[dimcf(0,xid)].m.xstart=xid;
        M[dimcf(0,xid)].y.xstart=xid+1;
    }
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END||Y_FREE_END&&X_FREE_END){
        M[dimcf(0,xid)].result().xend=xid;
        M[dimcf(0,xid)].result().yend=yid;
        *gbest=max2(*gbest,M[dimcf(0,xid)].result());
    }
}

int main(int argc,char** argv){
    int *gx_int,*gy_int;
    afg_unit *M,*GM;
    int xsize,ysize;
    int nthread,nblock;
    
    //common
    gscore_matrix_load();

    //讀取
    if(!load_file(&gx_int,&xsize,filename_x)){
        printf("讀不到 x 序列");
        exit(0);
    };
    if(!load_file(&gy_int,&ysize,filename_y)){
        printf("讀不到 y 序列");
        exit(0);
    }

    //宣告最佳解
    res_unit best,last;
    res_unit* gbest;
    hipMalloc(&gbest,sizeof(res_unit));
    hipMemcpy(gbest,&best,sizeof(res_unit),hipMemcpyHostToDevice);
    printf("*Remind: the interval start from 1, not 0\n");

    //挖記憶體
    M=new afg_unit[(xsize+2)*3];
    M+=3;
    hipMalloc(&GM, 3*(xsize+2)*sizeof(afg_unit));
    GM+=3;

    //分支
    switch(X_FREE_START+Y_FREE_START*2){
    case 0:
    case 1:
    case 3:
        M[dimcf(1,0)].m=0;
        hipMemcpy(GM-3, M-3, 3*(xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
        thread_assign(xsize+1,&nblock,&nthread);
        for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
            switch(X_FREE_START+Y_FREE_START*2){
            case 0:calculate_fixedStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,gbest,xsize,ysize);
                break;
            case 1:calculate_xfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,gbest,xsize,ysize);
                break;
            case 3:calculate_xyfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,gbest,xsize,ysize);
                break;
            }
            dim_move(GM,xsize+1);
        }
        break;
    case 2:
        M[dimcf(0,0)].m=0;
        M[dimcf(1,0)].m=0;
        M[dimcf(2,0)].m=0;
        hipMemcpy(GM-3, M-3, 3*(xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
        thread_assign(xsize,&nblock,&nthread);
        for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
            calculate_yfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,gbest,xsize,ysize);
            dim_move(GM+3,xsize);
        }
        break;
    }
    //印出結果
    hipMemcpy(&best,gbest,sizeof(res_unit),hipMemcpyDeviceToHost);//best score
    hipMemcpy(&last,GM+dimcf(0,xsize),sizeof(res_unit),hipMemcpyDeviceToHost);//last
    last.xend=xsize;
    last.yend=ysize;
    best=max2(best,last);
    show_best_and_output_file(best,xsize,ysize);
        
}

