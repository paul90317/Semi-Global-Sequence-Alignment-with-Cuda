#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>
#include "../common/afg_unit.cu"
#include "../common/score4.cu"
#include "../common/dim.cu"

#define BUF_SIZE_Y 10000
#define THREAD_SIZE 1024

using namespace std;

__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,char* x,char* y,int buf_mover,int index_y,res_unit* gbest,int xsize) {
    int t=threadIdx.x+blockDim.x*blockIdx.x;
    M[t+1].x=M1[t].to_x();
    M[t+1].y=M1[t+1].to_y();
    M[t+1].m=M2[t].to_m(x[t]==y[buf_mover-t]);
    if(t==0){
        M[t+1].x.start=index_y+1;
        M[t+1].m.start=index_y;
    }
    if(t+1==xsize){
        M[t+1].result().end=index_y-t;
        *gbest=max2(*gbest,M[t+1].result());
    }
    return;
}

__global__ void gmemset(int* m,int val) {
    int t=threadIdx.x+blockDim.x*blockIdx.x;
    m[t+1]=val;
}

int main(int argc,char** argv){
    FILE* file;
    char *x,*y;
    int *gx_int,*gy_int,*x_int,*y_int;
    afg_unit *M,*GM;
    int xsize,ysize,index_y;
    int nthread,nblock;
    if(argc!=3){
        cout<<"Follow format: command [x.txt] [y.txt]\n";
        return 0;
    }
    //common
    gscore_matrix_load();

    //讀取 x
    file=fopen(argv[1],"r");
    fseek(file,0,SEEK_END);
    xsize=ftell(file);
    fseek(file,0,SEEK_SET);
    x=new char[xsize+1];
    fgets(x,xsize+1,file);
    fclose(file);
    x_int=new int[xsize+1];
    x_int[0]=-1;
    for(int i=0;i<xsize;i++){
        x_int[i+1]=mapping_DNA(x[i]);
    }
    hipMalloc(&gx_int,sizeof(int)*(xsize+1));
    hipMemset(gx_int,-1,sizeof(int));
    hipMemcpy(gx_int+1,x_int,sizeof(int)*xsize,hipMemcpyHostToDevice);
    //讀取 y
    file=fopen(argv[2],"r");
    fseek(file,0,SEEK_END);
    ysize=ftell(file);
    fseek(file,0,SEEK_SET);
    y=new char[ysize+1];
    fgets(y,ysize+1,file);
    fclose(file);

    //初始化 cuda 參數
    nthread=min(xsize,THREAD_SIZE);
    nblock=xsize/THREAD_SIZE;
    if(xsize%THREAD_SIZE)nblock++;
    nsize=nblock*nthread;

    //動態規劃 M
    M=new afg_unit[nsize+1];
    M1=new afg_unit[nsize+1];
    M2=new afg_unit[nsize+1];

    //讀取 y buffer
    y=new char[BUF_SIZE_Y+1];
    file=fopen(argv[2],"r");
    fseek(file,0,SEEK_END);
    ysize=ftell(file);
    fseek(file,0,SEEK_SET);
    fgets (y , BUF_SIZE_Y+1 , file);// 讀 BUF_SIZE_Y 個

    //初始化 M
    buf_mover=0;
    index_y=0;

    M[0].m=0;
    M1[0].m=0;
    M2[0].m=0;

    //GPU COPY
    hipMalloc(&GM, (nsize+1)*sizeof(afg_unit));
    hipMalloc(&GM1, (nsize+1)*sizeof(afg_unit));
    hipMalloc(&GM2, (nsize+1)*sizeof(afg_unit));
    
    hipMemcpy(GM, M, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);
    hipMemcpy(GM1, M1, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);
    hipMemcpy(GM2, M2, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);

    hipMalloc(&gx, nsize*sizeof(char));
    hipMemset(gx, 0, nsize*sizeof(char));
    hipMemcpy(gx, x, xsize*sizeof(char), hipMemcpyHostToDevice);

    hipMalloc(&gy, (BUF_SIZE_Y+nsize+xsize-1)*sizeof(char));
    hipMemset(gy, 0, (BUF_SIZE_Y+nsize+xsize-1)*sizeof(char));
    oldgy=gy;
    nextgy=gy+BUF_SIZE_Y;
    gy=gy+nsize;
    hipMemcpy(gy, y, BUF_SIZE_Y*sizeof(char), hipMemcpyHostToDevice);//???

    //分段讀取並運算
    res_unit best;
    res_unit* gbest;
    hipMalloc(&gbest,sizeof(res_unit));
    hipMemcpy(gbest,&best,sizeof(res_unit),hipMemcpyHostToDevice);
    while(true){
        //可平行化運算
        while(buf_mover<BUF_SIZE_Y&&index_y<ysize){
            calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx,gy,buf_mover,index_y,gbest,xsize);
            hipMemcpy(GM2+1, GM1+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
            hipMemcpy(GM1+1, GM+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
            buf_mover++;
            index_y++;
        }
        
        //讀取 buffer
        if(fgets (y , BUF_SIZE_Y+1 , file)==NULL)break;
        //左移並寫入 buffer
        hipMemcpy(oldgy, nextgy, nsize*sizeof(char),hipMemcpyDeviceToDevice);
        buf_mover=0;
        hipMemcpy(gy, y, BUF_SIZE_Y*sizeof(char),hipMemcpyHostToDevice);
    };

    fclose(file);
    
    while (index_y<ysize+xsize-1)
    {
        calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx,gy,buf_mover,index_y,gbest,xsize);
        hipMemcpy(GM2+1, GM1+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
        hipMemcpy(GM1+1, GM+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
        buf_mover++;
        index_y++;
    }

    //取出結果
    hipMemcpy(&best,gbest,sizeof(res_unit),hipMemcpyDeviceToHost);//best score
    printf("%d %d %d",best.score,best.start,best.end);
}

