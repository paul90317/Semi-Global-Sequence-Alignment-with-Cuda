#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>
#include "afg_unit_gpu.h"

#define THREAD_SIZE 1024

using namespace std;

__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,char* x,char* y,int buf_mover,int* maxs,alm_node** bestalms,alm_node_control alc) {
    int t=threadIdx.x+blockDim.x*blockIdx.x;
    alm_node* tmp;

    if(x[t]==0||y[buf_mover-t]==0)return;

    tmp=alc.alm_node_alloc(t+1);
    M[t+1].stk_x=M1[t].alm_x(x[t],tmp);

    tmp=alc.alm_node_alloc(t+1);
    M[t+1].stk_y=M1[t+1].alm_y(y[buf_mover-t],tmp);

    tmp=alc.alm_node_alloc(t+1);
    M[t+1].stk_m=M2[t].alm_m(x[t],y[buf_mover-t],tmp);

    M[t+1].x=    M1[t]  .gto_x();
    M[t+1].y=    M1[t+1].gto_y();
    M[t+1].m=    M2[t]  .gto_m(x[t]==y[buf_mover-t]);

    bestalms[t+1]=max2sel(maxs[t+1],M[t+1].gresult(),bestalms[t+1],M[t+1].best_alm());
    maxs[t+1]=       max2(maxs[t+1],M[t+1].gresult());
    return;
}
void ptr_stk(alm_node* gstk,char* filename){
    alm_node *stk=nullptr,*tmp;
    FILE* fp;
    char buf[100];

    while(gstk){
        tmp=(alm_node*)malloc(sizeof(alm_node));
        hipMemcpy(tmp, gstk, sizeof(alm_node), hipMemcpyDeviceToHost);
        gstk=tmp->next;
        tmp->next=stk;
        stk=tmp;
    }

    fp=fopen(filename,"w");
    while(stk){
        sprintf(buf,"%c %c\n",stk->x,stk->y);
        fputs(buf,fp);
        stk=stk->next;
    }
    fclose(fp);
}

alm_node* cuda_alm_node(char x,char y,alm_node* next=nullptr){
    alm_node a,*b;
    a.x=x;
    a.y=y;
    a.next=next;
    hipMalloc(&b,sizeof(alm_node));
    hipMemcpy(b, &a, sizeof(alm_node), hipMemcpyHostToDevice);
    return b;
}

/*void debug(int * garr,int count){
    int *arr=new int[count];
    hipMemcpy(arr, garr, count*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<count;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");
}

void debug(afg_unit * garr,int count){
    afg_unit *arr=new afg_unit[count];
    hipMemcpy(arr, garr, count*sizeof(afg_unit), hipMemcpyDeviceToHost);
    for(int i=1;i<count;i++){
        printf("[r=%d,score=%d,%d,%d]:\n",i,arr[i].m,arr[i].x,arr[i].y);
        ptr_stk(arr[i].cbest_alm());
        printf("\n");
    }
}*/

int main(int argc,char** argv){
    FILE* file;
    char *x,*y,*gx,*gy;
    afg_unit *M,*M1,*M2,*GM,*GM1,*GM2;
    int xsize,buf_mover,tmp,nthread,nblock,nsize,*gmaxs,ysize;
    alm_node **gbestalms,*best_alm;

    if(argc!=4){
        cout<<"Follow format: command [x.txt] [y.txt] [out.txt]\n";
        return 0;
    }
    
    //讀取 x
    file=fopen(argv[1],"r");
    fseek(file,0,SEEK_END);
    xsize=ftell(file);
    x=new char[xsize+1];
    fseek(file,0,SEEK_SET);
    fgets(x,xsize+1,file);
    fclose(file);

    //初始化 cuda 參數
    nthread=min(xsize,THREAD_SIZE);
    nblock=xsize/THREAD_SIZE;
    if(xsize%THREAD_SIZE)nblock++;
    nsize=nblock*nthread;

    //動態規劃 M
    M=new afg_unit[nsize+1];
    M1=new afg_unit[nsize+1];
    M2=new afg_unit[nsize+1];

    //讀取 y buffer
    file=fopen(argv[2],"r");
    fseek(file,0,SEEK_END);
    ysize=ftell(file);
    y=new char[ysize+1];
    fseek(file,0,SEEK_SET);
    fgets (y , ysize+1 , file);
    fclose(file);

    //初始化 M
    M1[0].y=0;
    M2[0].y=0;

    M2[1].x=SCORE_G;
    M2[1].stk_x=cuda_alm_node(x[0],'-');
    M1[2].x=SCORE_G+SCORE_E;
    M1[2].stk_x=cuda_alm_node(x[1],'-',cuda_alm_node(x[0],'-'));

    M1[1].m=afg_unit::equal(x[0]==y[0]);
    M1[1].y=M2[1].to_y();
    M1[1].x=M2[0].to_x();
    
    M1[1].stk_m=cuda_alm_node(x[0],y[0]);
    M1[1].stk_x=cuda_alm_node(x[0],'-',cuda_alm_node('-',y[0]));
    M1[1].stk_y=cuda_alm_node('-',y[0],cuda_alm_node(x[0],'-'));

    buf_mover=1;

    //GPU COPY
    hipMalloc(&GM, (nsize+1)*sizeof(afg_unit));
    hipMemcpy(GM, M, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);

    hipMalloc(&GM1, (nsize+1)*sizeof(afg_unit));
    hipMemcpy(GM1, M1, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);

    hipMalloc(&GM2, (nsize+1)*sizeof(afg_unit));
    hipMemcpy(GM2, M2, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);

    hipMalloc(&gx, nsize*sizeof(char));
    hipMemset(gx, 0, nsize*sizeof(char));
    hipMemcpy(gx, x, xsize*sizeof(char), hipMemcpyHostToDevice);

    hipMalloc(&gy, (nsize+ysize+xsize-1)*sizeof(char));
    hipMemset(gy, 0, (nsize+ysize+xsize-1)*sizeof(char));
    gy=gy+nsize;
    hipMemcpy(gy, y, ysize*sizeof(char), hipMemcpyHostToDevice);

    tmp=NEG_INF;
    hipMalloc(&gmaxs, (nsize+1)*sizeof(int));
    hipMemcpy(gmaxs+xsize,&tmp,sizeof(int),hipMemcpyHostToDevice);

    hipMalloc(&gbestalms, (nsize+1)*sizeof(alm_node*));
    hipMemset(gbestalms+xsize,0,sizeof(alm_node*));

    //link list table
    alm_node_control alc(xsize);

    //可平行化運算
    while(buf_mover<ysize+xsize-1){
        calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx,gy,buf_mover,gmaxs,gbestalms,alc);
        buf_mover++;
        if(buf_mover==ysize+xsize-1){
            break;
        }
        hipMemcpy(GM2+1, GM1+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
        hipMemcpy(GM1+1, GM+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
    }

    //取出結果
    hipMemcpy(&tmp,gmaxs+xsize,sizeof(int),hipMemcpyDeviceToHost);
    printf("best's score: %d\n",tmp);
    hipMemcpy(&best_alm,gbestalms+xsize,sizeof(alm_node*),hipMemcpyDeviceToHost);
    ptr_stk(best_alm,argv[3]);
    printf("ALM saved finish!!\n");
}

