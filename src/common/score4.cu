#ifndef SCORE_H
#define SCORE_H
#include "afg_unit.cu"
int DNA_map[]={'A','T','C','G','-'};
__host__ inline int mapping_DNA(char c){
    switch(c){
    case 'A':
        return 0;
    case 'T':
        return 1;
    case 'C':
        return 2;
    case 'G':
        return 3;
    default:
        printf("mapping_DNA: DNA not found\n");
        exit(0);
    }
}
static datatype score_matrix[4][4]={
    {1,-1,-1,-1},
    {-1,1,-1,-1},
    {-1,-1,1,-1},
    {-1,-1,-1,1},
};
static datatype* gscore_matrix;
static bool loaded=false;
__host__ void gscore_matrix_load(){
    if(!loaded){
        hipMalloc(&gscore_matrix,16*sizeof(datatype));
        hipMemcpy(gscore_matrix,score_matrix,16*sizeof(datatype),hipMemcpyHostToDevice);//check this
        loaded=true;
    }
}
__device__ inline datatype gget_score(int i,int j){
    if(i==-1||j==-1)
        return -1;
    return gscore_matrix[4*i+j];
}

__host__ inline datatype cget_score(int i,int j){
    return score_matrix[i][j];
}
#endif