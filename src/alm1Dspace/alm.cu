#include <cstdlib>
#include <iostream>
#include <cstring>
#include "afg_controler.h"

#define THREAD_SIZE 1024

using namespace std;

char *x,*y;
res_unit res;
void dfs(FILE* file,afg_controler ac,int xl,int xr,int yl,int yr,bool xgap){
    int xs=(xr-xl+1),ys=(yr-yl+1);
    static char buf[1000];
    if(xs<=0){
        for(int i=yl-1;i<yr;i++){
            sprintf(buf,"- %c\n",y[i]);
            fputs(buf,file);
        }
        return;
    }
    if(ys<=0){
        for(int i=xl-1;i<xr;i++){
            sprintf(buf,"%c -\n",x[i]);
            fputs(buf,file);
        }
        return;
    }
    int ymid=(yl+yr)/2;
    res_unit tmp=ac.get_xmid(xl,xr,yl,yr,ymid,xgap);
    int xmid=tmp.start;
    if(tmp.is_backgap){//y 對應到 x 後的 gap
        dfs(file,ac,xl,xmid,yl,ymid-1,xgap);
        sprintf(buf,"- %c\n",y[ymid-1]);
        fputs(buf,file);
        dfs(file,ac,xmid+1,xr,ymid+1,yr,true);
    }else{
        dfs(file,ac,xl,xmid-1,yl,ymid-1,xgap);
        sprintf(buf,"%c %c\n",x[xmid-1],y[ymid-1]);
        fputs(buf,file);
        dfs(file,ac,xmid+1,xr,ymid+1,yr,false);
    }
    res=tmp;
}

int main(int argc,char** argv){
    FILE* file;
    int xsize,ysize;

    if(argc!=4){
        cout<<"Follow format: command [x.txt] [y.txt] [out.txt]\n";
        return 0;
    }
    
    //讀取 x
    file=fopen(argv[1],"r");
    fseek(file,0,SEEK_END);
    xsize=ftell(file);
    x=new char[xsize+1];
    fseek(file,0,SEEK_SET);
    fgets(x,xsize+1,file);
    fclose(file);

    //讀取 y buffer
    file=fopen(argv[2],"r");
    fseek(file,0,SEEK_END);
    ysize=ftell(file);
    y=new char[ysize+1];
    fseek(file,0,SEEK_SET);
    fgets (y , ysize+1 , file);
    fclose(file);

    //運算
    afg_controler ac(x,y,xsize,ysize);
    file=fopen(argv[3],"w");
    dfs(file,ac,1,xsize,1,ysize,false);
    fclose(file);
    printf("best score: %d\n",res.score);
}

