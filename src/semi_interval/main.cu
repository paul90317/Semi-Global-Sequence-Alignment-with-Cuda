#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>

#include "utils.cuh"
#include "afg_unit.cuh"
#include "config.cuh"
#include "layout.cuh"
#include "macro.cuh"
#include "test_time.h"


__global__ void calculate_yfreeStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count) {
    int tid=TID;
    int xid=tid+1;
    int yid=index_y-xid-1;
    if(xid<1||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(xid==1){
        M[dimcf(0,xid)].x.ystart=yid+1;
        M[dimcf(0,xid)].m.ystart=yid;
    }
#if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}
__global__ void calculate_xyfreeStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    res_unit zero(0,xid+1,xid,yid+1,yid);
    M[dimcf(0,xid)].x=max2(M[dimcf(1,xid-1)].to_x(),zero);
    M[dimcf(0,xid)].y=max2(M[dimcf(1,xid)].to_y(),zero);
    M[dimcf(0,xid)].m=max2(M[dimcf(2,xid-1)].to_m(x[xid],y[yid]),zero);
    #if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}
__global__ void calculate_fixedStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
#if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}
__global__ void calculate_xfreeStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(yid==0){
        M[dimcf(0,xid)].m=0;
    }
    if(yid==1){
        M[dimcf(0,xid)].m.xstart=xid;
        M[dimcf(0,xid)].y.xstart=xid+1;
    }
#if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}

int main(int argc,char** argv){
    int *gx_int,*gy_int;
    afg_unit *M,*GM;
    int xsize,ysize;
    int nthread,nblock;
    
    //common
    gscore_matrix_load();

    //讀取
    if(!load_file(&gx_int,&xsize,filename_x)){
        printf("讀不到 x 序列");
        exit(0);
    }
    std::cout<<filename_x<<": "<<xsize<<"\n";
    if(!load_file(&gy_int,&ysize,filename_y)){
        printf("讀不到 y 序列");
        exit(0);
    }
    std::cout<<filename_y<<": "<<ysize<<"\n";
    xsize=min(1000,xsize);
    ysize=min(10000,ysize);
    //宣告最佳解
    res_unit* g_best_stack;
    int* g_bs_count;
    hipMalloc(&g_best_stack,sizeof(res_unit)*BEST_STACK_SIZE);
    hipMalloc(&g_bs_count,sizeof(int));
    hipMemset(g_bs_count,0,sizeof(int));
    printf("*Remind: the interval start from 1, not 0\n");

    //挖記憶體
    M=new afg_unit[(xsize+2)*3];
    M+=3;
    hipMalloc(&GM, 3*(xsize+2)*sizeof(afg_unit));
    GM+=3;

    
    //分支
    time_start();
#if (START_MODE==2)
    M[dimcf(0,0)].m=0;
    M[dimcf(1,0)].m=0;
    M[dimcf(2,0)].m=0;
    hipMemcpy(GM-3, M-3, 3*(xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    thread_assign(xsize,&nblock,&nthread);
    for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
        calculate_yfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count);
        dim_move(GM+3,xsize);
    }
#else
    M[dimcf(1,0)].m=0;
    hipMemcpy(GM-3, M-3, 3*(xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    thread_assign(xsize+1,&nblock,&nthread);
    for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
    #if (START_MODE==0)
        calculate_fixedStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count);
    #elif (START_MODE==1)
        calculate_xfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count);
    #elif (START_MODE==3)
        calculate_xyfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count);
    #endif
        dim_move(GM,xsize+1);
    }
#endif
    time_end();

    //印出結果
#if (!X_FREE_END&&!Y_FREE_END)
    res_unit last;
    hipMemcpy(&last,GM+dimcf(0,xsize),sizeof(res_unit),hipMemcpyDeviceToHost);//last
    show_best_and_output_file(last,xsize,ysize);
#else
    res_unit*cbest_stack;
    int c_bs_count=interval_result_from_gup(&cbest_stack,g_best_stack,g_bs_count);
    show_best_and_output_file(cbest_stack,c_bs_count,xsize,ysize);
#endif
}

