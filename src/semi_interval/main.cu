#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>

#include "func.cuh"
#include "sequence.cuh"

#if END_MODE==0
__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,sequence x,sequence y,int offset_y,int offset_t)
#elif END_MODE==3
__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,sequence x,sequence y,int offset_y,int offset_t,res_unit_end* best_arr) 
#else
__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,sequence x,sequence y,int offset_y,int offset_t,res_unit_end* best_stack,int* bs_count,datatype* bscore) 
#endif
{
    int tid=TID+offset_t;
    int xid=tid;
    int yid=offset_y-tid;
    if(xid<0||yid<0||xid>x.size()||yid>y.size())return;
#if START_MODE==0
    M[xid].x=M1[xid-1].to_x();
    M[xid].y=M1[xid].to_y();
    M[xid].m=M2[xid-1].to_m(x.gget(xid),y.gget(yid));
#elif START_MODE==1
    M[xid].x=M1[xid-1].to_x();
    M[xid].y=M1[xid].to_y();
    M[xid].m=M2[xid-1].to_m(x.gget(xid),y.gget(yid));
    if(yid==0){
        M[xid].m=0;
        M[xid].m.xstart=xid+1;
        M[xid].y.xstart=xid+1;
    }
#elif START_MODE==2
    if(xid==0){
        M[xid].x.ystart=yid+1;
        M[xid].m.ystart=yid+1;
        return;
    }
    M[xid].x=M1[xid-1].to_x();
    M[xid].y=M1[xid].to_y();
    M[xid].m=M2[xid-1].to_m(x.gget(xid),y.gget(yid));
#elif START_MODE==3
    res_unit zero(0,xid+1,yid+1);
    M[xid].x=max2(M1[xid-1].to_x(),zero);
    M[xid].y=max2(M1[xid].to_y(),zero);
    M[xid].m=max2(M2[xid-1].to_m(x.gget(xid),y.gget(yid)),zero);
#endif
#if END_MODE==3
    res_unit_end now(M[xid].result(),xid,yid);
    best_arr[xid]=max2(now,best_arr[xid]);
#elif END_MODE>0
    if(xid==x.size()&&Y_FREE_END||yid==y.size()&&X_FREE_END){
        res_unit_end now(M[xid].result(),xid,yid);
        update_score(now,best_stack,bs_count,bscore);
    }
#endif
}
#define free_or_fixed(free) (free?"free":"fixed")
int main(int argc,char** argv){
    std::cout<<std::fixed<<std::setprecision(FLOAT_PRECISION)<<"\n";
    if(argc!=5){
        std::cout<<"error: follow format: semi_interval.exe <x.txt> <y.txt> <best interval.txt> <score.txt>\n";
        return 0;
    }
    afg_unit *GM,*GM1,*GM2;

    std::cout<<"semi-global-setting: src/headers/myconfig.h\n";
    std::cout<<" - x: ["<<free_or_fixed(X_FREE_START)<<", "<<free_or_fixed(X_FREE_END)<<"]\n";
    std::cout<<" - y: ["<<free_or_fixed(Y_FREE_START)<<", "<<free_or_fixed(Y_FREE_END)<<"]\n";
    if(!score::load(argv[4])){
        std::cout<<"error: can't load score matrix in "<<argv[4]<<"\n";
        exit(0);
    }else{
        std::cout<<"score matrix: "<<argv[4]<<"\n";
    }

    sequence x(argv[1]);
    sequence y(argv[2]);
    std::cout<<"sequence X: "<<argv[1]<<"\n";
    std::cout<<" - size: "<<x.size()<<"\n";
    std::cout<<"sequence Y: "<<argv[2]<<"\n";
    std::cout<<" - size: "<<y.size()<<"\n";
    
    //宣告最佳解
#if (X_FREE_END&&Y_FREE_END)
    res_unit_end* g_best_arr;
    hipMalloc(&g_best_arr,sizeof(res_unit_end)*(x.size()+1));
    assign_value2res(g_best_arr,NEG_INF,x.size()+1);
#elif START_MODE>0
    res_unit_end* g_best_stack;
    int* g_bs_count;
    hipMalloc(&g_best_stack,sizeof(res_unit_end)*BEST_STACK_SIZE);
    hipMalloc(&g_bs_count,sizeof(int));
    hipMemset(g_bs_count,0,sizeof(int));
    datatype* g_best_score;
    hipMalloc(&g_best_score,sizeof(datatype));
    assign_single(g_best_score,(datatype)NEG_INF);
#endif
    //挖記憶體
    {
        hipMalloc(&GM, (x.size()+2)*sizeof(afg_unit));
        hipMalloc(&GM1, (x.size()+2)*sizeof(afg_unit));
        hipMalloc(&GM2, (x.size()+2)*sizeof(afg_unit));
        afg_unit temp;
        assign_afg(GM,temp,x.size()+2);
        assign_afg(GM1,temp,x.size()+2);
        assign_afg(GM2,temp,x.size()+2);
        GM++;
        GM1++;
        GM2++;
        temp.m=0;
        assign_afg(GM,temp);
        assign_afg(GM1,temp);
        assign_afg(GM2,temp);
    }
    

    int nthread,nblock;
    int thread_needed,offset_t=0;
    //thread_assign(x.size()+1,&nblock,&nthread);
    mytime::start();
    for(int offset_y=1;offset_y-x.size()<=y.size();offset_y++){
        thread_needed=bound_assign(x.size(),y.size(),offset_y,&offset_t);
        thread_assign(thread_needed,&nblock,&nthread);
        #if (END_MODE==0)
            calculate _kernel(nblock,nthread)(GM,GM1,GM2,x,y,offset_y,offset_t);
        #elif (END_MODE==3)
            calculate _kernel(nblock,nthread)(GM,GM1,GM2,x,y,offset_y,offset_t,g_best_arr);
        #else
            calculate _kernel(nblock,nthread)(GM,GM1,GM2,x,y,offset_y,offset_t,g_best_stack,g_bs_count,g_best_score);
        #endif
        hipMemcpy(GM2,GM1,sizeof(afg_unit)*(x.size()+1),hipMemcpyDeviceToDevice);
        hipMemcpy(GM1,GM,sizeof(afg_unit)*(x.size()+1),hipMemcpyDeviceToDevice);
    }
    mytime::end();
    std::cout<<"[OUTPUT]\n";
    std::cout<<"best intervals: "<<argv[3]<<"\n";

    //印出結果
#if (!X_FREE_END&&!Y_FREE_END)
    afg_unit last;
    hipMemcpy(&last,GM+x.size(),sizeof(afg_unit),hipMemcpyDeviceToHost);//last
    std::cout<<"best score: "<<last.result().score<<"\n";
    show_best_and_output_file(argv[3],res_unit_end(last.result(),0,0),x.size(),y.size());
#elif (X_FREE_END&&Y_FREE_END)
    res_unit_end* cbests;
    datatype bestscore=interval_result_from_gup(&cbests,g_best_arr,x.size()+1);
    std::cout<<"best score: "<<bestscore<<"\n";
    show_best_and_output_file(argv[3],cbests,x.size()+1,x.size(),y.size(),bestscore);
#else
    datatype ctmp_bscore;
    hipMemcpy(&ctmp_bscore,g_best_score,sizeof(datatype),hipMemcpyDeviceToHost);
    std::cout<<"best score: "<<ctmp_bscore<<"\n";
    res_unit_end*cbest_stack;
    int c_bs_count=interval_result_from_gup(&cbest_stack,g_best_stack,g_bs_count);
    show_best_and_output_file(argv[3],cbest_stack,c_bs_count,x.size(),y.size(),ctmp_bscore);
#endif
}

