#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>

#include "utils.cuh"
#include "afg_unit.cuh"
#include "config.h"
#include "layout.cuh"
#include "macro.cuh"
#include "test_time.h"

__global__ void calculate_yfreeStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count,datatype* bscore) {
    int tid=TID;
    int xid=tid+1;
    int yid=index_y-xid-1;
    if(xid<1||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(xid==1){
        M[dimcf(0,xid)].x.ystart=yid+1;
        M[dimcf(0,xid)].m.ystart=yid;
    }
#if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count,bscore);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}
__global__ void calculate_xyfreeStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count,datatype* bscore) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    res_unit zero(0,xid+1,xid,yid+1,yid);
    M[dimcf(0,xid)].x=max2(M[dimcf(1,xid-1)].to_x(),zero);
    M[dimcf(0,xid)].y=max2(M[dimcf(1,xid)].to_y(),zero);
    M[dimcf(0,xid)].m=max2(M[dimcf(2,xid-1)].to_m(x[xid],y[yid]),zero);
    #if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count,bscore);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}
__global__ void calculate_fixedStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count,datatype* bscore) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
#if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count,bscore);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}
__global__ void calculate_xfreeStart(afg_unit* M,int* x,int* y,int index_y,int xsize,int ysize,res_unit* best_stack,int* bs_count,datatype* bscore) {
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
    M[dimcf(0,xid)].x=M[dimcf(1,xid-1)].to_x();
    M[dimcf(0,xid)].y=M[dimcf(1,xid)].to_y();
    M[dimcf(0,xid)].m=M[dimcf(2,xid-1)].to_m(x[xid],y[yid]);
    if(yid==0){
        M[dimcf(0,xid)].m=0;
    }
    if(yid==1){
        M[dimcf(0,xid)].m.xstart=xid;
        M[dimcf(0,xid)].y.xstart=xid+1;
    }
#if (Y_FREE_END || X_FREE_END)
#if !(Y_FREE_END&&X_FREE_END)
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
#endif
        res_unit& now=M[dimcf(0,xid)].result();
        update_score(now,xid,yid,best_stack,bs_count,bscore);
#if !(Y_FREE_END&&X_FREE_END)
    }
#endif
#endif
}

int main(int argc,char** argv){
    int *gx_int,*gy_int;
    afg_unit *M,*GM;
    int xsize,ysize;
    int nthread,nblock;
    
    //common
    gscore_matrix_load();

    //讀取
    if(!load_file(&gx_int,&xsize,filename_x)){
        printf("讀不到 x 序列");
        exit(0);
    }
    std::cout<<"X sequence: "<<filename_x<<" , Global interval=[1, "<<xsize<<"]\n";
    if(!load_file(&gy_int,&ysize,filename_y)){
        printf("讀不到 y 序列");
        exit(0);
    }
    std::cout<<"Y sequence: "<<filename_y<<" , Global interval=[1, "<<ysize<<"]\n";
    //宣告最佳解
    res_unit* g_best_stack;
    int* g_bs_count;
    hipMalloc(&g_best_stack,sizeof(res_unit)*BEST_STACK_SIZE);
    hipMalloc(&g_bs_count,sizeof(int));
    hipMemset(g_bs_count,0,sizeof(int));

    //挖記憶體
    M=new afg_unit[(xsize+2)*3];
    M+=3;
    hipMalloc(&GM, 3*(xsize+2)*sizeof(afg_unit));
    GM+=3;

    datatype* g_best_score;
    hipMalloc(&g_best_score,sizeof(datatype));
    assign_single(g_best_score,(datatype)NEG_INF);
    
    //分支
    time_start();
#if (START_MODE==2)
    M[dimcf(0,0)].m=0;
    M[dimcf(1,0)].m=0;
    M[dimcf(2,0)].m=0;
    hipMemcpy(GM-3, M-3, 3*(xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    thread_assign(xsize,&nblock,&nthread);
    for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
        calculate_yfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count,g_best_score);
        dim_move(GM+3,xsize);
    }
#else
    M[dimcf(1,0)].m=0;
    hipMemcpy(GM-3, M-3, 3*(xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    thread_assign(xsize+1,&nblock,&nthread);
    for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
    #if (START_MODE==0)
        calculate_fixedStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count,g_best_score);
    #elif (START_MODE==1)
        calculate_xfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count,g_best_score);
    #elif (START_MODE==3)
        calculate_xyfreeStart<<<nblock,nthread>>>(GM,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count,g_best_score);
    #endif
        dim_move(GM,xsize+1);
    }
#endif
    time_end();
    std::cout<<"Best interval saved in: "<<filename_best_score_interval<<"\n\n";
    //印出結果
#if (!X_FREE_END&&!Y_FREE_END)
    res_unit last;
    hipMemcpy(&last,GM+dimcf(0,xsize),sizeof(res_unit),hipMemcpyDeviceToHost);//last
    std::cout<<"Best score: "<<last.score<<"\n";
    show_best_and_output_file(last,xsize,ysize);
#else
    datatype ctmp_bscore;
    hipMemcpy(&ctmp_bscore,g_best_score,sizeof(datatype),hipMemcpyDeviceToHost);
    std::cout<<"Best score: "<<ctmp_bscore<<"\n";
    res_unit*cbest_stack;
    int c_bs_count=interval_result_from_gup(&cbest_stack,g_best_stack,g_bs_count);
    show_best_and_output_file(cbest_stack,c_bs_count,xsize,ysize,ctmp_bscore);
#endif
}

