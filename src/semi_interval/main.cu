#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>

#include "utils.cuh"
#include "afg_unit.cuh"
#include "config.h"
#include "layout.cuh"
#include "macro.cuh"
#include "test_time.h"

#if END_MODE==0
__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,byte* x,byte* y,int index_y,int xsize,int ysize)
#elif END_MODE==3
__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,byte* x,byte* y,int index_y,int xsize,int ysize,res_unit_end* best_arr) 
#else
__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,byte* x,byte* y,int index_y,int xsize,int ysize,res_unit_end* best_stack,int* bs_count,datatype* bscore) 
#endif
{
    int xid=TID;
    int yid=index_y-xid-1;
    if(xid<0||yid<0||xid>xsize||yid>ysize)return;
#if START_MODE==0
    M[xid].x=M1[xid-1].to_x();
    M[xid].y=M1[xid].to_y();
    M[xid].m=M2[xid-1].to_m(x[xid],y[yid]);
#elif START_MODE==1
    M[xid].x=M1[xid-1].to_x();
    M[xid].y=M1[xid].to_y();
    M[xid].m=M2[xid-1].to_m(x[xid],y[yid]);
    if(yid==0){
        M[xid].m=0;
    }
    if(yid==1){
        M[xid].m.xstart=xid;
        M[xid].y.xstart=xid+1;
    }
#elif START_MODE==2
    if(xid==0)return;
    M[xid].x=M1[xid-1].to_x();
    M[xid].y=M1[xid].to_y();
    M[xid].m=M2[xid-1].to_m(x[xid],y[yid]);
    if(xid==1){
        M[xid].x.ystart=yid+1;
        M[xid].m.ystart=yid;
    }
#elif START_MODE==3
    res_unit zero(0,xid+1,yid+1);
    M[xid].x=max2(M1[xid-1].to_x(),zero);
    M[xid].y=max2(M1[xid].to_y(),zero);
    M[xid].m=max2(M2[xid-1].to_m(x[xid],y[yid]),zero);
#endif
#if END_MODE==3
    res_unit_end now(M[xid].result(),xid,yid);
    best_arr[xid]=max2(now,best_arr[xid]);
#elif END_MODE>0
    if(xid==xsize&&Y_FREE_END||yid==ysize&&X_FREE_END){
        res_unit_end now(M[xid].result(),xid,yid);
        update_score(now,best_stack,bs_count,bscore);
    }
#endif
}

int main(int argc,char** argv){
    byte *gx_int,*gy_int;
    afg_unit *M,*M1,*M2,*GM,*GM1,*GM2;
    int xsize,ysize;
    int nthread,nblock;
    if(argc!=4){
        std::cout<<"follow format: semi_interval.exe [x.txt] [t.txt] [best interval.txt]\n";
        return 0;
    }
    //common
    gscore_matrix_load();

    //讀取
    if(!load_file(&gx_int,&xsize,argv[1])){
        printf("讀不到 x 序列");
        exit(0);
    }
    std::cout<<"X sequence: "<<argv[1]<<" , Global interval=[1, "<<xsize<<"]\n";
    if(!load_file(&gy_int,&ysize,argv[2])){
        printf("讀不到 y 序列");
        exit(0);
    }
    std::cout<<"Y sequence: "<<argv[2]<<" , Global interval=[1, "<<ysize<<"]\n";
    
    //宣告最佳解
#if (!X_FREE_END&&!Y_FREE_END)
#elif (X_FREE_END&&Y_FREE_END)
    res_unit_end* g_best_arr;
    hipMalloc(&g_best_arr,sizeof(res_unit_end)*(xsize+1));
    assign_arr(g_best_arr,NEG_INF,xsize+1);
#else
    res_unit_end* g_best_stack;
    int* g_bs_count;
    hipMalloc(&g_best_stack,sizeof(res_unit_end)*BEST_STACK_SIZE);
    hipMalloc(&g_bs_count,sizeof(int));
    hipMemset(g_bs_count,0,sizeof(int));
    datatype* g_best_score;
    hipMalloc(&g_best_score,sizeof(datatype));
    assign_single(g_best_score,(datatype)NEG_INF);
#endif
    //挖記憶體
    M=new afg_unit[xsize+2];
    M1=new afg_unit[xsize+2];
    M2=new afg_unit[xsize+2];
    M++;
    M1++;
    M2++;
    hipMalloc(&GM, (xsize+2)*sizeof(afg_unit));
    hipMalloc(&GM1, (xsize+2)*sizeof(afg_unit));
    hipMalloc(&GM2, (xsize+2)*sizeof(afg_unit));
    GM++;
    GM1++;
    GM2++;
    M[0].m=0;
    M1[0].m=0;
    M2[0].m=0;
    hipMemcpy(GM-1, M-1, (xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    hipMemcpy(GM1-1, M1-1, (xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    hipMemcpy(GM2-1, M2-1, (xsize+2)*sizeof(afg_unit), hipMemcpyHostToDevice);
    thread_assign(xsize+1,&nblock,&nthread);

    //分支
    time_start();
    for(int idy=2;Y_NOT_END(idy,xsize,ysize);idy++){
        #if (END_MODE==0)
            calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx_int,gy_int,idy,xsize,ysize);
        #elif (END_MODE==3)
            calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx_int,gy_int,idy,xsize,ysize,g_best_arr);
        #else
            calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx_int,gy_int,idy,xsize,ysize,g_best_stack,g_bs_count,g_best_score);
        #endif
        hipMemcpy(GM2,GM1,sizeof(afg_unit)*(xsize+1),hipMemcpyDeviceToDevice);
        hipMemcpy(GM1,GM,sizeof(afg_unit)*(xsize+1),hipMemcpyDeviceToDevice);
    }
    time_end();
    std::cout<<"Best interval saved in: "<<argv[3]<<"\n\n";

    //印出結果
#if (!X_FREE_END&&!Y_FREE_END)
    afg_unit last;
    hipMemcpy(&last,GM+xsize,sizeof(afg_unit),hipMemcpyDeviceToHost);//last
    std::cout<<"Best score: "<<last.result().score<<"\n";
    show_best_and_output_file(argv[3],res_unit_end(last.result(),0,0),xsize,ysize);
#elif (X_FREE_END&&Y_FREE_END)
    res_unit_end* cbests;
    datatype bestscore=interval_result_from_gup(&cbests,g_best_arr,xsize+1);
    std::cout<<"Best score: "<<bestscore<<"\n";
    show_best_and_output_file(argv[3],cbests,xsize+1,xsize,ysize,bestscore);
#else
    datatype ctmp_bscore;
    hipMemcpy(&ctmp_bscore,g_best_score,sizeof(datatype),hipMemcpyDeviceToHost);
    std::cout<<"Best score: "<<ctmp_bscore<<"\n";
    res_unit_end*cbest_stack;
    int c_bs_count=interval_result_from_gup(&cbest_stack,g_best_stack,g_bs_count);
    show_best_and_output_file(argv[3],cbest_stack,c_bs_count,xsize,ysize,ctmp_bscore);
#endif
}

