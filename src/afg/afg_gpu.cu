#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>
#include "afg_unit_gpu.h"

#define BUF_SIZE_Y 10000
#define THREAD_SIZE 1024

using namespace std;

__global__ void calculate(afg_unit* M,afg_unit* M1,afg_unit* M2,char* x,char* y,int buf_mover,int index_y,int* maxs,int* init_idy,int* last_idy) {
    int t=threadIdx.x+blockDim.x*blockIdx.x;
    M[t+1].x=M1[t].gto_x();
    M[t+1].y=M1[t+1].gto_y();
    M[t+1].m=M2[t].gto_m(x[t]==y[buf_mover-t]);
    M[t+1].xstart=M1[t].start_x(index_y-t,t);
    M[t+1].ystart=M1[t+1].start_y();
    M[t+1].mstart=M2[t].start_m(index_y-t,t);
    init_idy[t+1]=M[t+1].bests_idy(maxs[t+1],init_idy[t+1]);
    last_idy[t+1]=M[t+1].bests_lastidy(maxs[t+1],last_idy[t+1],index_y-t);
    maxs[t+1]=max2(maxs[t+1],M[t+1].gresult());
    return;
}

__global__ void gmemset(int* m,int val) {
    int t=threadIdx.x+blockDim.x*blockIdx.x;
    m[t+1]=val;
}

int main(int argc,char** argv){
    FILE* file;
    char *x,*y,*gx,*gy,*oldgy,*nextgy;
    afg_unit *M,*M1,*M2,*GM,*GM1,*GM2;
    int xsize,buf_mover,tmp,nthread,nblock,nsize,index_y,*gmaxs,*init_idy,*last_idy,ysize;

    if(argc!=3){
        cout<<"Follow format: command [x.txt] [y.txt]\n";
        return 0;
    }

    //讀取 x
    file=fopen(argv[1],"r");
    fseek(file,0,SEEK_END);
    xsize=ftell(file);
    fseek(file,0,SEEK_SET);
    x=new char[xsize+1];
    fgets(x,xsize+1,file);
    fclose(file);

    //初始化 cuda 參數
    nthread=min(xsize,THREAD_SIZE);
    nblock=xsize/THREAD_SIZE;
    if(xsize%THREAD_SIZE)nblock++;
    nsize=nblock*nthread;

    //動態規劃 M
    M=new afg_unit[nsize+1];
    M1=new afg_unit[nsize+1];
    M2=new afg_unit[nsize+1];

    //讀取 y buffer
    y=new char[BUF_SIZE_Y+1];
    file=fopen(argv[2],"r");
    fseek(file,0,SEEK_END);
    ysize=ftell(file);
    fseek(file,0,SEEK_SET);
    fgets (y , BUF_SIZE_Y+1 , file);// 讀 BUF_SIZE_Y 個

    //初始化 M
    buf_mover=0;
    index_y=1;

    M1[0].y=0;
    M2[0].y=0;

    M2[1].x=SCORE_G;
    M1[2].x=SCORE_G+SCORE_E;

    M1[1].m=afg_unit::equal(x[0]==y[buf_mover++]);
    M1[1].y=M2[1].to_y();
    M1[1].x=M2[0].to_x();

    M1[1].mstart=0;
    M1[1].xstart=0;
    M1[1].ystart=0;

    //GPU COPY
    hipMalloc(&GM, (nsize+1)*sizeof(afg_unit));
    hipMalloc(&GM1, (nsize+1)*sizeof(afg_unit));
    hipMalloc(&GM2, (nsize+1)*sizeof(afg_unit));
    
    hipMemcpy(GM, M, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);
    hipMemcpy(GM1, M1, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);
    hipMemcpy(GM2, M2, (nsize+1)*sizeof(afg_unit), hipMemcpyHostToDevice);

    hipMalloc(&gx, nsize*sizeof(char));
    hipMemset(gx, 0, nsize*sizeof(char));
    hipMemcpy(gx, x, xsize*sizeof(char), hipMemcpyHostToDevice);

    hipMalloc(&gy, (BUF_SIZE_Y+nsize+xsize-1)*sizeof(char));
    hipMemset(gy, 0, (BUF_SIZE_Y+nsize+xsize-1)*sizeof(char));
    oldgy=gy;
    nextgy=gy+BUF_SIZE_Y;
    gy=gy+nsize;
    hipMemcpy(gy, y, BUF_SIZE_Y*sizeof(char), hipMemcpyHostToDevice);//???

    hipMalloc(&gmaxs, (nsize+1)*sizeof(int));
    tmp=NEG_INF;
    hipMemcpy(gmaxs+xsize,&tmp,sizeof(int),hipMemcpyHostToDevice);

    hipMalloc(&init_idy, (nsize+1)*sizeof(int));
    hipMalloc(&last_idy, (nsize+1)*sizeof(int));

    //分段讀取並運算
    while(true){
        //可平行化運算
        while(buf_mover<BUF_SIZE_Y&&index_y<ysize){
            calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx,gy,buf_mover,index_y,gmaxs,init_idy,last_idy);
            hipMemcpy(GM2+1, GM1+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
            hipMemcpy(GM1+1, GM+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
            buf_mover++;
            index_y++;
        }
        
        //讀取 buffer
        if(fgets (y , BUF_SIZE_Y+1 , file)==NULL)break;
        //左移並寫入 buffer
        hipMemcpy(oldgy, nextgy, nsize*sizeof(char),hipMemcpyDeviceToDevice);
        buf_mover=0;
        hipMemcpy(gy, y, BUF_SIZE_Y*sizeof(char),hipMemcpyHostToDevice);
    };

    fclose(file);
    
    while (index_y<ysize+xsize-1)
    {
        calculate<<<nblock,nthread>>>(GM,GM1,GM2,gx,gy,buf_mover,index_y,gmaxs,init_idy,last_idy);
        hipMemcpy(GM2+1, GM1+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
        hipMemcpy(GM1+1, GM+1, nsize*sizeof(afg_unit),hipMemcpyDeviceToDevice);
        buf_mover++;
        index_y++;
    }

    //取出結果
    hipMemcpy(&tmp,gmaxs+xsize,sizeof(int),hipMemcpyDeviceToHost);//best score
    printf("%d ",tmp);
    hipMemcpy(&tmp,init_idy+xsize,sizeof(int),hipMemcpyDeviceToHost);//inital index
    printf("%d ",tmp);
    hipMemcpy(&tmp,last_idy+xsize,sizeof(int),hipMemcpyDeviceToHost);//last index
    printf("%d",tmp);
}

