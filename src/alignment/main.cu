#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>
#include <iomanip>
#include "afg_controller.cuh"
#include "file.cuh"
#include "test_time.h"
#include "check_alm.h"
#include "alm_controller/alm_controller.cuh"

namespace dfs{
    byte *x_int,*y_int;
    FILE* file;
    afg_controller afg_c;
    alm_controller alm_c;
    datatype bscore;
    void dfs(int xl,int xr,int yl,int yr,bool xgap){
        int xs=(xr-xl+1),ys=(yr-yl+1);
        if(xs<=0){
            for(int i=yl;i<=yr;i++){
                print_alm(file,'-',to_Char(y_int[i]));
            }
            return;
        }
        if(ys<=0){
            for(int i=xl;i<=xr;i++){
                print_alm(file,to_Char(x_int[i]),'-');
            }
            return;
        }
        if(xs<ALM_END_POINT_SIZE&&ys<ALM_END_POINT_SIZE){
            bscore=alm_c.cal_out_trace_back(file,xl,xr,yl,yr,xgap);
            return;
        }
        int ymid=(yl+yr)/2;
        res_unit tmp=afg_c.get_xmid(xl,xr,yl,yr,ymid,xgap);
        int xmid=tmp.xmid; 
        if(tmp.is_xbackgap){//y 對應到 x 後的 gap
            dfs(xl,xmid,yl,ymid-1,xgap);
            print_alm(file,'-',to_Char(y_int[ymid]));;
            dfs(xmid+1,xr,ymid+1,yr,true);
        }else{
            dfs(xl,xmid-1,yl,ymid-1,xgap);
            print_alm(file,to_Char(x_int[xmid]),to_Char(y_int[ymid]));
            dfs(xmid+1,xr,ymid+1,yr,false);
        }
        bscore=tmp.score;
    }
}

bool is_same(datatype a,datatype b){
    if(typeid(datatype)==typeid(float)){
        return abs(a-b)<=ERROR_FLOAT;
    }
    if(typeid(datatype)==typeid(double)){
        //std::cout<<std::fixed<<std::setprecision(5)<<a<<" "<<b<<"\n";
        return abs(a-b)<=ERROR_FLOAT;
    }
    if(typeid(datatype)==typeid(long double)){
        return abs(a-b)<=ERROR_FLOAT;
    }
    return a==b;
    
}

int main(int argc,char** argv){
    byte *gx,*gy;
    if(argc!=6){
        std::cout<<"Error: follow format => alignment.exe [x.txt] [y.txt] [best interval.txt] [score.txt] [alignment.txt]\n";
        return 0;
    }
    //common
    if(!score::load(argv[4])){
        std::cout<<"Error: can't load score matrix in "<<argv[4]<<"\n";
        exit(0);
    }else{
        std::cout<<"loaded score matrix in "<<argv[4]<<"\n";
    }

    //讀取 best interval
    datatype score;
    int xl,xr,yl,yr;

    //讀取 best interval
    if(!load_best_interval(argv[3],&score,&xl,&xr,&yl,&yr)){
        std::cout<<"Error: can't open best interval!!!\n";
        exit(0);
    }
    std::cout<<"Load semi interval from "<<argv[3]<<" , Index=" <<DEFAULT_INTERVAL_INDEX<<", Score="<<score<<"\n";
    int xsz=xr-xl+1;
    int ysz=yr-yl+1;
    
    //讀取 x
    if(!load_file(&gx,&dfs::x_int,argv[1],xl,xr)){
        std::cout<<"Error: can't read x sequence!!!\n";
        exit(0);
    }
    std::cout<<"X sequence: "<<argv[1]<<" , Semi interval=["<<xl<<", "<<xr<<"]\n";

    //讀取 y
    if(!load_file(&gy,&dfs::y_int,argv[2],yl,yr)){
        std::cout<<"Error: can't read y sequence!!!\n";
        exit(0);
    }
    std::cout<<"Y sequence: "<<argv[2]<<" , Semi interval=["<<yl<<", "<<yr<<"]\n";

    //運算
    dfs::afg_c=afg_controller(gx,gy,xsz);
    dfs::alm_c=alm_controller(gx,gy,dfs::x_int,dfs::y_int);
    dfs::file=fopen(argv[5],"w");
    time_start();
    dfs::dfs(1,xsz,1,ysz,false);
    time_end();
    fclose(dfs::file);
    std::cout<<"Best score: "<<dfs::bscore<<"\n";
    
    datatype chk_score;
    bool match=check_alm(argv[5],dfs::x_int,dfs::y_int,&chk_score);
    if(!match){
        std::cout<<"Error: the alignment don't match original sequences!!!\n";
    }else{
        std::cout<<"The score of alignment "<<argv[5]<<" is "<<chk_score<<"\n";
    }
    if(!is_same(dfs::bscore,score)||!is_same(dfs::bscore,chk_score)){
        std::cout<<"Error: the score is not the same!!!\n";
    }
}

