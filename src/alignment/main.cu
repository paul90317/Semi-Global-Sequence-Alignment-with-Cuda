#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cstring>
#include <iomanip>
#include "sequence.cuh"
#include "alm_controller/alm_controller.cuh"
#include "afg_controller.cuh"

namespace dfs{
    FILE* file;
    afg_controller *afg_c;
    alm_controller *alm_c;
    datatype bscore;
    void dfs(sequence x,sequence y, bool xgap){
        if(x.size()<=0){
            for(int i=1;i<=y.size();i++){
                print_alm(file,'-',score::Char_map[y.cget(i)]);
            }
            return;
        }
        if(y.size()<=0){
            for(int i=1;i<=x.size();i++){
                print_alm(file,score::Char_map[x.cget(i)],'-');
            }
            return;
        }
        if(x.size()<ALM_END_POINT_SIZE&&y.size()<ALM_END_POINT_SIZE){
            bscore=alm_c->cal_out_trace_back(file,x,y,xgap);
            return;
        }
        int ymid=(1+y.size())/2;
        res_unit tmp=afg_c->get_xmid(x,y,ymid,xgap);
        int xmid=tmp.xmid;
        if(tmp.is_xbackgap){//y 對應到 x 後的 gap
            dfs(x.subseq(1,xmid),y.subseq(1,ymid-1),xgap);
            print_alm(file,'-',score::Char_map[y.cget(ymid)]);;
            dfs(x.subseq(xmid+1,x.size()-xmid),y.subseq(ymid+1,y.size()-ymid),true);
        }else{
            dfs(x.subseq(1,xmid-1),y.subseq(1,ymid-1),xgap);
            print_alm(file,score::Char_map[x.cget(xmid)],score::Char_map[y.cget(ymid)]);;
            dfs(x.subseq(xmid+1,x.size()-xmid),y.subseq(ymid+1,y.size()-ymid),false);
        }
        bscore=tmp.score;
    }
}

bool is_same(datatype a,datatype b){
if (typeid(datatype)==typeid(float) || typeid(datatype)==typeid(double) || typeid(datatype)==typeid(long double))
    return abs(a-b)<=FLOAT_ERROR;
else
    return a==b;
}

int main(int argc,char** argv){
    std::cout<<"\n";
    if(argc!=6){
        std::cout<<"error: follow format: alignment.exe <x.txt> <y.txt> <best interval.txt> <score.txt> <alignment.txt>\n";
        return 0;
    }
    //common
    if(!score::load(argv[4])){
        std::cout<<"error: can't load score matrix in "<<argv[4]<<"\n";
        exit(0);
    }else{
        std::cout<<"score matrix: "<<argv[4]<<"\n";
    }
    sequence x,y;
    datatype interval_score;
    //讀取 best interval
    {
        int xl,xr,yl,yr;

        //讀取 best interval
        if(!load_best_interval(argv[3],&interval_score,&xl,&xr,&yl,&yr)){
            std::cout<<"error: can't open best interval file\n";
            exit(0);
        }
        std::cout<<"interval: "<<argv[3]<<"\n";
        std::cout<<" - index: " <<DEFAULT_INTERVAL_INDEX<<"\n";
        std::cout<<" - score: " <<interval_score<<"\n";
        x=sequence(argv[1]).subseq(xl,xr-xl+1);
        y=sequence(argv[2]).subseq(yl,yr-yl+1);
        std::cout<<" - sequence X: "<<argv[1]<<"\n";
        std::cout<<" -  - interval: ["<<xl<<", "<<xr<<"]\n";
        std::cout<<" - sequence Y: "<<argv[2]<<"\n";
        std::cout<<" -  - interval: ["<<yl<<", "<<yr<<"]\n";
    }
    

    //運算
    dfs::afg_c=new afg_controller(x.size());
    dfs::alm_c=new alm_controller();
    dfs::file=fopen(argv[5],"w");
    mytime::start();
    dfs::dfs(x,y,false);
    mytime::end();
    fclose(dfs::file);
    std::cout<<"[OUTPUT]\n";
    std::cout<<"best score: "<<dfs::bscore<<"\n";
    
    datatype chk_score;
    bool match=check_alm(argv[5],x,y,&chk_score);
    if(!match){
        std::cout<<"error: the alignment don't match original sequences\n";
        exit(0);
    }else{
        std::cout<<"alignment: "<<argv[5]<<"\n";
        std::cout<<" - score: "<<chk_score<<"\n";
    }
    if(!is_same(dfs::bscore,interval_score)||!is_same(dfs::bscore,chk_score)){
        std::cout<<"error: the score is not the same\n";
    }
}

