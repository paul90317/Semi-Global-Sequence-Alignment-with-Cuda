#include "hip/hip_runtime.h"

#include <iostream>
#include "test_time.h"
#include "utils.cuh"

#define COUNT 1000000
#define TIME 100000
__global__ void _move(int* a,int *b,int count){
    int tid=TID;
    if(tid>=count)return;
    a[tid]=b[tid];
}
void move(int* a,int* b,int count){
    int nb,nt;
    thread_assign(count,&nb,&nt);
    _move _kernel(nb,nt)(a,b,count);
}

int main(){
    int *a,*b;
    hipMalloc(&a,sizeof(int)*COUNT);
    hipMalloc(&b,sizeof(int)*COUNT);
    time_start();
    for(int i=0;i<TIME;i++){
        move(a,b,COUNT);
        //hipMemcpy(a,b,sizeof(int)*COUNT,hipMemcpyDeviceToDevice);
    }
    time_end();
}